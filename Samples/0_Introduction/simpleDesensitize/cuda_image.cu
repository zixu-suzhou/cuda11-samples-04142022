#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif

// Includes, system
#include <math.h>
// #include <npp.h>
// #include <nppi.h>
#include <stdio.h>

#include <cassert>
// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check
#include <pthread.h>
#include <semaphore.h>

#include <cstdint>

#include "cuda_image.h"
#include <vector>

#define LOGCUDA(...) printf("CUDA\t" __VA_ARGS__)
#define MAX_STREAM_COUNT 1

typedef struct cuda_dev_ctrl {
  /*decide which debug information you want present. refer to
   * cuda_debug-info-value defined above. */
  int32_t debug_info;
  pthread_mutex_t mutex;
  // gpu total number
  uint32_t device_count;
  // stream count
  uint32_t stream_count;
} cuda_dev_ctrl_t;


static uint32_t is_init = 0;
static cuda_dev_ctrl_t g_cuda_dev_ctrl;
static hipStream_t cuda_stream_array[MAX_STREAM_COUNT];
///////////////////////////////////////////global
///kernel//////////////////////////////////////////////
__global__ void dev_yuv422_single_mask(size_t width, size_t height, uint8_t *yuv_in_out,
    size_t *mat, size_t num_mat) {

  size_t scalex, scaley, fx0, fy0;
  size_t i, j; //j=width, i=hight
  size_t threadid;
  size_t blockid;
  size_t *p;

  blockid = blockIdx.x + blockIdx.y * gridDim.x;
  threadid = blockid * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) +
             threadIdx.x;

  j = (threadid * 2) % width;
  i = ((threadid * 2) - j) / width;


  for(size_t l = 0; l < num_mat; l++){
    p = (size_t*)(mat + l * 4);
    fx0 = *p;
    fy0 = *(p + 1);
    scalex = *(p + 2);
    scaley = *(p + 3);
    if((j > fx0) 
        && (j < fx0 + scalex)
        && (i > fy0)
        && (i < fy0 + scaley)
      ){
      yuv_in_out[threadid * 4] = 0; //erase Y
      yuv_in_out[(threadid * 4) + 2] = 0; //erase pixel 2 ->Y
    }
  }
}

__global__ void dev_yuvn12_single_mask(size_t width, size_t height, uint8_t *yuyv_in_out, 
    size_t *mat, size_t num_mat) {

  size_t i, j;
  j = blockIdx.x * blockDim.x + threadIdx.x;
  i = blockIdx.y * blockDim.y + threadIdx.y;

  size_t scalex, scaley, fx0, fy0;
  size_t *p;
  for(size_t l = 0; l < num_mat; l++){
    p = (size_t*)(mat + l * 4);
    fx0 = *p;
    fy0 = *(p + 1);
    scalex = *(p + 2);
    scaley = *(p + 3);
    if((j > fx0) 
        && (j < fx0 + scalex)
        && (i > fy0)
        && (i < fy0 + scaley)
      ){
      yuyv_in_out[i * width + j] = 0; //y = black
    }
  }
}

///////////////////////////////////////////host
///funcs//////////////////////////////////////////////

static int host_yuv422_single_mask(uint8_t *yuvbuffer,
                              size_t width, size_t height, cuda_mat_t *mat, size_t num_mat,
                              hipStream_t stream) {
  size_t iw, ih;
  if (NULL == yuvbuffer) {
    LOGCUDA("%s invalid param\n", __func__);
    return -1;
  }
  if(num_mat > 100){
    LOGCUDA("illegal mat num %zu", num_mat);
    return -1;
  }
  size_t *gpu_mat = (size_t*)cuda_malloc(num_mat * sizeof(size_t) * 4);
  cuda_memcpy(gpu_mat, mat, num_mat * sizeof(size_t) * 4, CUDA_HOST_TO_DEV);

  /* 8 bytes contains 4 pixels */
  iw = width / 2;
  ih = height;
  dim3 dimBlock(64, 8);
  dim3 dimGrid(iw / dimBlock.x, ih / dimBlock.y);

  dev_yuv422_single_mask<<<dimGrid, dimBlock, 0, stream>>>(width, height, yuvbuffer, (size_t *)gpu_mat, num_mat);

  cuda_free(gpu_mat);

  return 0;
}


static int host_yuvn12_single_mask(uint8_t *yuvbuffer,
                              size_t width, size_t height, cuda_mat_t *mat, size_t num_mat,
                              hipStream_t stream) {
  size_t iw, ih;
  if (NULL == yuvbuffer) {
    LOGCUDA("%s invalid param\n", __func__);
    return -1;
  }
  if(num_mat > 100){
    LOGCUDA("illegal mat num %zu", num_mat);
    return -1;
  }
  size_t *gpu_mat = (size_t*)cuda_malloc(num_mat * sizeof(size_t) * 4);
  cuda_memcpy(gpu_mat, mat, num_mat * sizeof(size_t) * 4, CUDA_HOST_TO_DEV);

  iw = width;
  ih = height;

  dim3 dimBlock(64, 8);
  dim3 dimGrid((iw + dimBlock.x - 1) / dimBlock.x,
               (ih + dimBlock.y - 1) / dimBlock.y);
  dev_yuvn12_single_mask<<<dimGrid, dimBlock, 0, stream>>>(width, height, yuvbuffer, (size_t *)gpu_mat, num_mat);
  cuda_free(gpu_mat);

  return 0;
}


extern "C" int cuda_YUVMASK(cuda_image_handle_t *cuda_handle_image, cuda_mat_t *mat, size_t num_mat) {
  int error = -1;
  uint32_t stream_idx;
  cuda_image_cmd_e image_cmd;
  size_t width, height;
  uint8_t *srcbuffer;
  std::vector<unsigned int> results;


  if (NULL == cuda_handle_image) {
    LOGCUDA("%s: invalid param\n", __func__);
    goto failed;
  }
  srcbuffer = (uint8_t *)cuda_handle_image->imagein->plan_buffer[0];
  width = cuda_handle_image->imagein->width;
  height = cuda_handle_image->imagein->height;
  image_cmd = cuda_handle_image->image_cmd;

  stream_idx = cuda_handle_image->stream_idx;
  if (stream_idx > (MAX_STREAM_COUNT - 1)) {
    LOGCUDA("%s: exceed valid stream count\n", __func__);
    goto failed;
  }

  switch (image_cmd) {
    case CUDA_IMG_YUV422:
      error = host_yuv422_single_mask(srcbuffer, width, height, mat, num_mat, 
          cuda_stream_array[stream_idx]);
      if (error) {
        LOGCUDA("%s: host_yuv422_single_mask err\n", __func__);
        goto failed;
      }
      break;

    case CUDA_IMG_UVY422:
      break;

    case CUDA_IMG_YUV420PITCH:
      break;

    case CUDA_IMG_YUV420SPPITCH:
      break;

    case CUDA_IMG_YUV420SP:
      error = host_yuvn12_single_mask(srcbuffer, width, height, mat, num_mat, 
          cuda_stream_array[stream_idx]);
      if (error) {
        LOGCUDA("%s: host_yuvn12_single_mask err\n", __func__);
        goto failed;
      }
      break;

    case CUDA_IMG_I420:
      break;

    default:
      LOGCUDA("%s: invalid image cmd\n", __func__);
      goto failed;
  }

  return 0;

failed:
  return -1;
}

///////////////////////////////////////////host api----init or
///config//////////////////////////////////////////////
extern "C" int cuda_get_dev_count(uint32_t *dev_count) {
  hipError_t error;

  if (NULL == dev_count) {
    LOGCUDA("%s: invalid param\n", __func__);
    return -1;
  }

  error = hipGetDeviceCount((int *)dev_count);
  if (error != hipSuccess) {
    LOGCUDA("%s: hipGetDeviceCount returned %d\n-> %s\n", __func__, (int)error,
            hipGetErrorString(error));
    return -1;
  }

  return 0;
}

extern "C" int cuda_get_dev_info(cuda_dev_info_t *cuda_dev_info) {
  int i;
  hipError_t error;
  uint32_t deviceCount;
  int attrValue = 0;
  int driverVersion = 0, runtimeVersion = 0;

  if (NULL == cuda_dev_info) {
    LOGCUDA("%s: invalid param\n", __func__);
    return -1;
  }

  error = hipGetDeviceCount((int *)&deviceCount);
  if (error != hipSuccess) {
    LOGCUDA("%s: hipGetDeviceCount returned %d\n-> %s\n", __func__, (int)error,
            hipGetErrorString(error));
    return -1;
  }
  cuda_dev_info->device_count = deviceCount;

  for (i = 0; i < deviceCount; i++) {
    checkCudaErrors(hipSetDevice(i));
    
    checkCudaErrors(hipDeviceGetAttribute(&attrValue, hipDeviceAttributeUnifiedAddressing, i));
    cuda_dev_info->cuda_dev_property[i].device_count_with_unified_memory = attrValue;

    checkCudaErrors(hipDriverGetVersion(&driverVersion));
    checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
    cuda_dev_info->driver_version = driverVersion;
    cuda_dev_info->runtime_version = runtimeVersion;
  }

  return 0;
}

extern "C" int cuda_dev_init(void) {
  if (is_init) return 0;

  hipError_t error;
  int deviceCount;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  pcuda_dev_ctrl->debug_info = CUDA_DEBUG_NULL;

  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    LOGCUDA("%s: hipGetDeviceCount returned %d\n-> %s\n", __func__, (int)error,
            hipGetErrorString(error));
    return -1;
  }

  if (!deviceCount) {
    LOGCUDA("%s: no gpu...\n", __func__);
    return -1;
  }

  pthread_mutex_init(&pcuda_dev_ctrl->mutex, NULL);
  pcuda_dev_ctrl->device_count = deviceCount;
  pcuda_dev_ctrl->stream_count = 0;

  for (uint32_t i = 0; i < MAX_STREAM_COUNT; i++) {
    cuda_stream_create(i);
  }

  is_init = 1;

  return 0;
}

extern "C" int cuda_dev_uninit(void) {
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  pcuda_dev_ctrl->stream_count = 0;

  return 0;
}

extern "C" int cuda_dev_check(uint32_t *devidx) {
  hipError_t error;

  error = hipGetDevice((int *)devidx);
  if (error != hipSuccess) {
    LOGCUDA("%s: err, returned %d\n-> %s\n", __func__, (int)error,
            hipGetErrorString(error));
    return -1;
  }

  return 0;
}

extern "C" int cuda_dev_select(uint32_t devidx) {
  hipError_t error;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  if (devidx > (pcuda_dev_ctrl->device_count - 1)) {
    LOGCUDA("%s: invalid devidx...\n", __func__);
    return -1;
  }

  error = hipSetDevice(devidx);
  if (error != hipSuccess) {
    LOGCUDA("%s: err, returned %d\n-> %s\n", __func__, (int)error,
            hipGetErrorString(error));
    return -1;
  }

  return 0;
}

extern "C" int cuda_stream_create(uint32_t streamidx) {
  unsigned int cnt;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  pthread_mutex_lock(&pcuda_dev_ctrl->mutex);
  cnt = pcuda_dev_ctrl->stream_count;
  if (cnt > (MAX_STREAM_COUNT - 1)) {
    LOGCUDA("%s: exceed valid stream count\n", __func__);
    pthread_mutex_unlock(&pcuda_dev_ctrl->mutex);
    return -1;
  }
  checkCudaErrors(hipStreamCreateWithFlags(&cuda_stream_array[cnt], hipStreamNonBlocking));
  cnt++;
  pcuda_dev_ctrl->stream_count = cnt;
  pthread_mutex_unlock(&pcuda_dev_ctrl->mutex);

  return 0;
}

extern "C" int cuda_stream_destroy(void) {
  unsigned int i, cnt;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  cnt = pcuda_dev_ctrl->stream_count;
  for (i = 0; i < cnt; i++) {
    checkCudaErrors(hipStreamDestroy(cuda_stream_array[i]));
  }

  return 0;
}

extern "C" int cuda_stream_sync(uint32_t streamidx) {
  hipError_t error;
  unsigned int cnt;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  cnt = pcuda_dev_ctrl->stream_count;
  if (streamidx > (cnt - 1)) {
    LOGCUDA("%s: exceed valid stream count\n", __func__);
    goto failed;
  }

  error = hipStreamSynchronize(cuda_stream_array[streamidx]);
  if (error) {
    LOGCUDA("%s:\n err: %d %s\n\n", __func__, error, hipGetErrorString(error));
    goto failed;
  }

  return 0;
failed:
  return -1;
}

extern "C" void *cuda_malloc_unified(size_t size, cuda_memory_type_e type) {
  void *buffer = NULL;

  switch (type) {
    case CUDA_HOST:
      checkCudaErrors(hipHostMalloc((void **)&buffer, size));
      if (buffer) {
        return buffer;
      }
      break;
    case CUDA_ATTACH_HOST:
      checkCudaErrors(
          hipHostAlloc((void **)&buffer, size, hipMemAttachHost));
      if (buffer) {
        return buffer;
      }
      break;
    case CUDA_ATTACH_GOLABLE:
    default:
      checkCudaErrors(hipHostAlloc((void **)&buffer, size, hipMemAttachHost));
      if (buffer) {
        return buffer;
      }
      break;
  }

  return NULL;
}

extern "C" void *cuda_malloc(size_t size) {
  void *buffer = NULL;

  checkCudaErrors(hipMalloc((void **)&buffer, size));
  if (buffer) {
    return buffer;
  }

  return NULL;
}

extern "C" void *cuda_memcpy(void *dst, void *src, size_t size,
                             cuda_memcpy_direct_e dir) {
  switch (dir) {
    case CUDA_DEV_TO_HOST:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost,
                                      cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));
      break;
    case CUDA_HOST_TO_DEV:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice,
                                      cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));
      break;
    case CUDA_DEV_TO_DEV:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice,
                                      cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));
      break;
    default:
      return NULL;
  }

  return dst;
}

extern "C" void *cuda_memcpy_async(void *dst, void *src, size_t size,
                                   cuda_memcpy_direct_e dir, uint32_t idx) {
  unsigned int cnt;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  cnt = pcuda_dev_ctrl->stream_count;
  if (idx > (cnt - 1)) {
    LOGCUDA("%s: exceed valid stream count\n", __func__);
    return NULL;
  }

  switch (dir) {
    case CUDA_DEV_TO_HOST:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost,
                                      cuda_stream_array[idx]));
      break;
    case CUDA_HOST_TO_DEV:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice,
                                      cuda_stream_array[idx]));
      break;
    case CUDA_DEV_TO_DEV:
      checkCudaErrors(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice,
                                      cuda_stream_array[idx]));
      break;
    default:
      return NULL;
  }

  return dst;
}

extern "C" void *cuda_memcpy2D(void *dst, size_t dpitch, const void *src,
                               size_t spitch, size_t width, size_t height,
                               cuda_memcpy_direct_e dir) {
  switch (dir) {
    case CUDA_DEV_TO_HOST:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyDeviceToHost,
                                        cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));
      break;
    case CUDA_HOST_TO_DEV:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyHostToDevice,
                                        cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));
      break;
    case CUDA_DEV_TO_DEV:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyDeviceToDevice,
                                        cuda_stream_array[0]));
      checkCudaErrors(hipStreamSynchronize(cuda_stream_array[0]));  
      break;
    default:
      return NULL;
  }

  return dst;
}

extern "C" void *cuda_memcpy2D_async(void *dst, size_t dpitch, const void *src,
                                     size_t spitch, size_t width, size_t height,
                                     cuda_memcpy_direct_e dir, uint32_t idx) {
  unsigned int cnt;
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  cnt = pcuda_dev_ctrl->stream_count;
  if (idx > (cnt - 1)) {
    LOGCUDA("%s: exceed valid stream count\n", __func__);
    return NULL;
  }

  switch (dir) {
    case CUDA_DEV_TO_HOST:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyDeviceToHost,
                                        cuda_stream_array[idx]));
      break;
    case CUDA_HOST_TO_DEV:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyHostToDevice,
                                        cuda_stream_array[idx]));
      break;
    case CUDA_DEV_TO_DEV:
      checkCudaErrors(hipMemcpy2DAsync(dst, dpitch, src, spitch, width, height,
                                        hipMemcpyDeviceToDevice,
                                        cuda_stream_array[idx]));
      break;
    default:
      return NULL;
  }

  return dst;
}

extern "C" void cuda_free(void *buffer) {
  if (buffer) {
    checkCudaErrors(hipFree(buffer));
  }
}

extern "C" void cuda_debug(int32_t debg_info) {
  cuda_dev_ctrl_t *pcuda_dev_ctrl = &g_cuda_dev_ctrl;

  pcuda_dev_ctrl->debug_info = debg_info;
}
